#include "hip/hip_runtime.h"
#include "vectorSumCUDA.h"
#include <stdlib.h>
#include <stdio.h>
#include <cmath>
__global__ void vectorSumCUDA(double *aDev, double *bDev, double *cDev) {
	//Have to think in 3D, this CUDA function will do a triple nested loop with
	//reduction. NOTE no for statements anywhere!
	const int atid = blockIdx.x ; //number from 0-1024, similar to i loop
	const int btid = blockIdx.y ; //number from 0-1024, similar to j loop
	//The tid below is a local thread in the block, each block has 1024 threads
	//so this will range from 0-1024
	const int tid = (threadIdx.x * blockDim.x) + threadIdx.y;
	//atomicAdd is CUDA's add function to take care of race conditions
	atomicAdd(&cDev[tid],  (sinf(aDev[atid]) * sinf(bDev[btid]) * pow(cosf(aDev[btid]),5)) * sinf(tid));
	//Bad CUDA implementation below, this would work if called on one block but would
	//be very slow
	/*for (int i = 0;i < 1024;i++) {
		for (int j = 0;j < 1024;j++)
			atomicAdd(&cDev[tid], (sinf(aDev[i]) * sinf(bDev[j]) * pow(cosf(aDev[j]),5)) * sinf(tid));
	}*/

}

