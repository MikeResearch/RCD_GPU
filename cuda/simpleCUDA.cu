#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simplestDeviceFunction(int *dev) {
	//threadIdx.x is the ID of the thread in x direction, can have 3D
	int tid = threadIdx.x;
	//tid in this program will be anywhere from 0 to size - 1, all executing at once
	dev[tid] = tid;
}
int main() {
	int size = 64;
	int a[size];
	//dim3 - special CUDA type to declare how many blocks and threads you want
        dim3 numBlocks(1,1,1); //BLOCKS, not threads
        dim3 threadsPerBlock(size,1);    //How many threads are in each block
	int *a_device;
	//Memory management, have to declare space on device
	hipError_t gpucheck;
	gpucheck = hipMalloc(&a_device, size * sizeof(int));
	if (gpucheck != hipSuccess) {
		std::cout << "Error allocating memory on GPU. No GPU?" << std::endl;
		exit(1);
	}
	//Call CUDA function on graphics card - function name, resources,parameters
	simplestDeviceFunction<<< numBlocks, threadsPerBlock >>>(a_device);
	//After work is done, copy result from GPU back to CPU
	hipMemcpy(a, a_device, size*sizeof(int), hipMemcpyDeviceToHost);
	//What is in a?
	for (int i = 0;i < size;i++)
		std::cout << a[i] << std::endl;
}
