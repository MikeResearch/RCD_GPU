#include "hip/hip_runtime.h"
#include "wrapVectorSum.h"
#include "vectorSumCUDA.h"
#include <iostream>
void wrapVectorSum(double *a, double *b, double *c, int size) {
	double *a_device, *b_device, *c_device;
	//Allocate space on GPU for our work
	hipError_t gpucheck;
        gpucheck = hipMalloc(&a_device, size * sizeof(double)); 
	if (gpucheck != hipSuccess) {
                std::cout << "Error allocating memory on GPU. No GPU?" << std::endl;
                exit(1);
        };
	hipMalloc(&b_device, size * sizeof(double));
	hipMalloc(&c_device, size * sizeof(double)); // size * sizeof(int));
	//Copy a and b to GPU, we don't copy c since it's the result
	hipMemcpy(a_device, a, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(b_device, b, size * sizeof(double), hipMemcpyHostToDevice);
	//CUDA max threads per block is 1024, this is CUDA limitation!
	int maxThreadsPerBlock = 1024;
	int sqrtMaxThreads = (int) sqrt(maxThreadsPerBlock);
	int totalNumBlocks = (size / maxThreadsPerBlock); 
	//int remainder = size % maxBlockSize;
	//int sqr = (int) sqrt(totalNumBlocks);
	//dim3 numBlocks(1,1,1); //bad implementation
	dim3 numBlocks(size,size,1); //good 3D implementaiton

	//Here we get into potential block issues if you're bigger than 1024
	dim3 threadsPerBlock(sqrtMaxThreads,sqrtMaxThreads);   
	std::cout << "calling a total of " << size * size << " blocks with " << sqrtMaxThreads << " x " << sqrtMaxThreads << " threads " << std::endl;
	//Call function in CUDA, <<< >>> is how many resources you're using
	//() is parameter list
	vectorSumCUDA <<< numBlocks, threadsPerBlock >>>(a_device, b_device, c_device);
	//Copy result from GPU back to CPU
	hipMemcpy(c, c_device , size * sizeof(double),hipMemcpyDeviceToHost);
	//No memory leaks, all mallocs should be freed !
	hipFree(a_device); hipFree(b_device); hipFree(c_device);
}

